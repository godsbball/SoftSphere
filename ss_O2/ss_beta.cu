#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include <stdio.h>

#include <stdlib.h>
#include <math.h>
#include <time.h>

// for memset
#include  <cstring>

#define SCALE 2.328306436e-10
static unsigned int ux, uy;
double xor64( )
{
    unsigned int t = (ux^(ux<<8));
    ux = uy;
    uy = (uy^(uy>>22))^(t^(t>>9));
    return ((double) uy) * SCALE; 
}        

float h, h2;
static int L = 10;


#include "global.h"
#include "host.c"
#include "ovito_plot.c"

#include "cell.c"
#include "device.c"


int main(int argc, char** argv)
{

    int label;
    sscanf (argv[1],"%d",&label);

    char file_name[50];

    FILE *fin0;
    sprintf(file_name,"par.dat");
    fin0 = fopen(file_name, "r");

    int N_p;
    fscanf(fin0,"%d\n", &N_p);
    fscanf(fin0,"%f\n", &pf0);
    fscanf(fin0,"%f\n", &mesh_fac);
    fscanf(fin0,"%d\n", &tobs);
    float frac_total_part;
    fscanf(fin0,"%f\n", &frac_total_part);
    fscanf(fin0,"%f\n", &h);

    fclose(fin0);


    N = 1 << N_p;

// 	srand( 300101985 );
    int seed = (int)time(NULL);
//     int seed = 124095837;

    srand( seed );
    int ii;
    for (ii=0; ii<100; ++ii) ux = rand();
    for (ii=0; ii<100; ++ii) uy = rand();
    


//#################################################################################
//host_variables_initialization
    int NO_BYTES = N*sizeof(float);
    
    x_h = (float*)malloc(NO_BYTES);
    y_h = (float*)malloc(NO_BYTES);

    px_h = (float*)malloc(NO_BYTES);
    py_h = (float*)malloc(NO_BYTES);

    _2r_h = (float*)malloc(NO_BYTES);

//cell
    L_mesh = L/(float)mesh_fac;

    L2_mesh = L_mesh*L_mesh;
    cell_num = L2_mesh;
    num_particle_per_cell = (int)(frac_total_part*N);

    int NO_BYTES_cell = cell_num*num_particle_per_cell*sizeof(int);
    int NO_BYTES_cell_ind = (cell_num)*sizeof(int);

//     int *cell_ind_h;
//     int *cell_h;
//
//     cell_h = (int*)malloc(NO_BYTES_cell);
//     cell_ind_h = (int*)malloc(NO_BYTES_cell_ind);
//
//     for (ii=0; ii < cell_num; ++ii)
//         cell_ind_h[ii] = 0;




//#################################################################################
//initial conditions importation (host)

    sprintf(file_name,"./input/c_000.dat");
    FILE *fin2;
    fin2 = fopen(file_name, "r");

    for (ii=0; ii < N; ++ii)
        fscanf(fin2,"%f %f %f\n", &x_h[ii], &y_h[ii], &_2r_h[ii]);

    fclose(fin2);


    sprintf(file_name,"./input/par_000.dat");
    FILE *fin3;
    fin3 = fopen(file_name, "r");

    fscanf(fin3,"%d %f", &ii, &amp_fac);

    fclose(fin3);

//#################################################################################
//initial conditions generation (host)

//     r_dist();
//
//     amp_fac = 1.e-3;
//     V_hard_sphere();
//
//
//     while( V_hs/( (float)(L*L)/pow(amp_fac,2) ) < pf0 ){
//         amp_fac += 0.0001;
//     }
//
//     pf =V_hs/( (float)(L*L)/pow(amp_fac,2));
    for (ii=0; ii<N; ++ii){

//         x_h[ii] = (float)(L)*xor64();
//         y_h[ii] = (float)(L)*xor64();

        px_h[ii] =  1e-2*(2.*xor64() - 1);
        py_h[ii] =  1e-2*(2.*xor64() - 1);

    }

    V_hard_sphere();
    pf = V_hs/( (float)(L*L)/pow(amp_fac,2));

    h2 = 0.5*h;

    printf("\n#############################\n");
    printf("N = %d \n",N);
    printf("tobs = %d \n", tobs);
    printf("h = %g \n", h);
    printf("seed = %d \n", seed);
    printf("#############################\n");

    printf("\ncell_num: %d   cell_num*num_particle_per_cell: %d L_mesh: %d\n", cell_num, cell_num*num_particle_per_cell, L_mesh);
    printf("num_particle_per_cell: %d mesh_fac: %g\n\n", num_particle_per_cell, mesh_fac);

    printf("amp_fac: %g  pf: %g\n\n", amp_fac, pf);
//#################################################################################
//device_variables_initialization

    float *x_d, *y_d;
    float *_2r_d;

    int *cell_ind_d;
    int *cell_d;

    float *px_d, *py_d;
    float *fx_d, *fy_d;

    float *u_d;


    hipMalloc((float **)&x_d, NO_BYTES);
    hipMemcpy(x_d,x_h, NO_BYTES, hipMemcpyHostToDevice);

    hipMalloc((float **)&y_d, NO_BYTES);
    hipMemcpy(y_d,y_h, NO_BYTES, hipMemcpyHostToDevice);

    hipMalloc((float **)&_2r_d, NO_BYTES);
    hipMemcpy(_2r_d,_2r_h,NO_BYTES, hipMemcpyHostToDevice);

    hipMalloc((float **)&px_d, NO_BYTES);
    hipMemcpy(px_d,px_h, NO_BYTES, hipMemcpyHostToDevice);

    hipMalloc((float **)&py_d, NO_BYTES);
    hipMemcpy(py_d,py_h, NO_BYTES, hipMemcpyHostToDevice);

//     hipMalloc((float **)&px_d, NO_BYTES);
//     hipMemset(px_d, 0, NO_BYTES);
//
//     hipMalloc((float **)&py_d, NO_BYTES);
//     hipMemset(py_d, 0, NO_BYTES);



    hipMalloc((int **)&cell_d, NO_BYTES_cell);
    hipMalloc((int **)&cell_ind_d, NO_BYTES_cell_ind);

    hipMemset(cell_d, 0, NO_BYTES_cell);
    hipMemset(cell_ind_d, 0, NO_BYTES_cell_ind);

    hipMalloc((float **)&fx_d, NO_BYTES);
    hipMalloc((float **)&fy_d, NO_BYTES);
    hipMemset(fx_d, 0, NO_BYTES);
    hipMemset(fy_d, 0, NO_BYTES);

    hipMalloc((float **)&u_d, NO_BYTES);
    hipMemset(u_d, 0, NO_BYTES);


//#################################################################################
//device_structure_definition
    
    device_structure_definition();

    printf("\ngrid: %d,  block: %d,  c1*grid*block = %d,  ct,ct2 = %d,%d,  BSF = %d\n", NumBlock,block_size,ct*NumBlock*block_size,ct,ct2,BSF);



    hipMemcpyToSymbol(HIP_SYMBOL(dct1), &ct, sizeof(int));//device parameter
    hipMemcpyToSymbol(HIP_SYMBOL(dct2), &ct2, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(mesh_d), &mesh_fac, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(L_mesh_d), &L_mesh, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(num_particle_per_cell_d), &num_particle_per_cell, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(d_h), &h, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_h2), &h2, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(L_d), &L, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(L_mesh_d), &L_mesh, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(L2_mesh_d), &L2_mesh, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(cell_num_d), &cell_num, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(amp_fac_d), &amp_fac, sizeof(float));

//     float temp;
//
//     temp = 1.1;
//     hipMemcpyToSymbol(HIP_SYMBOL(f_inc), &temp, sizeof(float));
//     temp = 0.5;
//     hipMemcpyToSymbol(HIP_SYMBOL(f_dec), &temp, sizeof(float));
//     temp = 0.99;
//     hipMemcpyToSymbol(HIP_SYMBOL(f_alpha), &temp, sizeof(float));
//     temp = 0.2;
//     hipMemcpyToSymbol(HIP_SYMBOL(alpha), &temp, sizeof(float));
//     hipMemcpyToSymbol(HIP_SYMBOL(alpha_start), &temp, sizeof(float));
//
//     ii = 0;
//     hipMemcpyToSymbol(HIP_SYMBOL(ii_bac), &ii, sizeof(int));
//
//     temp = 30*h;
//     hipMemcpyToSymbol(HIP_SYMBOL(hmax), &temp, sizeof(float));



//#################################################################################
//backup_variables_definition

    int temp_array_byte_size = sizeof(float)*NumBlock;

    float *d_temp;
    hipMalloc((void**)&d_temp,temp_array_byte_size);
    hipMemset(d_temp, 0., temp_array_byte_size);

    float *d_temp2;
    hipMalloc((void**)&d_temp2,temp_array_byte_size);
    hipMemset(d_temp2, 0., temp_array_byte_size);

    float *d_temp3;
    hipMalloc((void**)&d_temp3,temp_array_byte_size);
    hipMemset(d_temp3, 0., temp_array_byte_size);


    float *dE;
    hipMalloc((void**)&dE, temp_array_byte_size);
    hipMemset(dE, 0., temp_array_byte_size);

    float *dVec;
    hipMalloc((void**)&dVec, temp_array_byte_size);
    hipMemset(dVec, 0., temp_array_byte_size);


//#################################################################################
//#################################################################################
//run_device
//     hipError_t cudaStatus;

//     int jj;
    float energy_bac0, energy_bac1, energy_bac2;
    energy_bac0 = energy_bac1 = energy_bac2 = 1.;


    for (ii=0; ii<tobs; ++ii){

        EvolveP<<<NumBlock,block_size>>>(x_d, y_d, px_d, py_d, fx_d, fy_d, cell_ind_d, cell_d, _2r_d, u_d);
        hipDeviceSynchronize();

        EvolveQ<<<NumBlock, block_size>>>(x_d, y_d, px_d, py_d);
        hipDeviceSynchronize();

        initializeCellInd<<<NumBlock, block_size>>>(cell_ind_d);
        hipDeviceSynchronize();
        initializeCells<<<NumBlock,block_size>>>(x_d, y_d, cell_ind_d, cell_d);
        hipDeviceSynchronize();

        EvolveP<<<NumBlock,block_size>>>(x_d, y_d, px_d, py_d, fx_d, fy_d, cell_ind_d, cell_d, _2r_d, u_d);
        hipDeviceSynchronize();


        if(ii%1000==0){
            Energy_reduction<<<NumBlock,block_size>>>(px_d, py_d, u_d, d_temp, d_temp2);
            Energy_FinalReduction<<<1,BSF>>>(d_temp, d_temp2, dE);
            hipDeviceSynchronize();

            hipMemcpy(&energy_bac0, &dE[0], sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&energy_bac1, &dE[1], sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&energy_bac2, &dE[2], sizeof(float), hipMemcpyDeviceToHost);

            printf("\ntemperature: %.3g\t potential_e/N: %.3g\t tot_e: %.3g\t ii: %d\n", energy_bac0/(float)N, energy_bac1/(float)N, energy_bac2, ii);
        }

    }

    Energy_reduction<<<NumBlock,block_size>>>(px_d, py_d, u_d, d_temp, d_temp2);
    Energy_FinalReduction<<<1,BSF>>>(d_temp, d_temp2, dE);
    hipDeviceSynchronize();

//#################################################################################
//Copy from device to host

    int *cell_ind_h_bac = (int*)malloc(NO_BYTES_cell_ind);
    hipMemcpy(cell_ind_h_bac, cell_ind_d, NO_BYTES_cell_ind, hipMemcpyDeviceToHost);

    int *cell_h_bac = (int*)malloc(NO_BYTES_cell);
    hipMemcpy(cell_h_bac, cell_d, NO_BYTES_cell, hipMemcpyDeviceToHost);

    hipMemcpy(x_h, x_d, NO_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(y_h, y_d, NO_BYTES, hipMemcpyDeviceToHost);

    hipMemcpy(px_h, px_d, NO_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(py_h, py_d, NO_BYTES, hipMemcpyDeviceToHost);

    float *u_h = (float*)malloc(NO_BYTES);
    hipMemcpy(u_h, u_d, NO_BYTES, hipMemcpyDeviceToHost);

    float *dh_cpy = (float*)malloc(temp_array_byte_size);
    hipMemcpy(dh_cpy, dE, temp_array_byte_size, hipMemcpyDeviceToHost);
    printf("\nenergy_device: %g\n", dh_cpy[2]);


    hipFree(cell_ind_d);
    hipFree(cell_d);

    hipFree(x_d);
    hipFree(y_d);

    hipFree(px_d);
    hipFree(py_d);

    hipFree(fx_d);
    hipFree(fy_d);

    hipFree(dE);
    hipFree(u_d);

    hipFree(dVec);

    float E_check = 0;
    for (ii=0; ii < N; ++ii){


        E_check += 0.5*(px_h[ii]*px_h[ii] + py_h[ii]*py_h[ii]);
        E_check += u_h[ii];
    }

    printf("energy_host: %g\n", E_check);


//#################################################################################
    printf("\n\n############### device task concluded\n\n");
//plot and check


    sprintf(file_name,"test_h.dat");
    FILE *fout0;
    fout0 = fopen(file_name, "w");

    for (ii=0; ii < N; ++ii)
        fprintf(fout0,"%g %g %g\n", px_h[ii], py_h[ii], u_h[ii]);

    fclose(fout0);
    ovito_plot(0);

    return 0;
}
